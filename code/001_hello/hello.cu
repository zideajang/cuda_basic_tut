#include<stdio.h> // 引入标准输入输出库，用于printf函数
#include<hip/hip_runtime.h>  // 引入CUDA运行时库，包含了CUDA相关的函数和类型定义

// device d_
// host h_
// 上面两行是注释，通常用来标记变量是分配在设备端（GPU）还是主机端（CPU）

__global__ // 这是一个CUDA关键字，表示其后的函数是一个可以在GPU上执行的“核函数”
void kernel() // 定义了一个名为kernel的函数，它将在GPU上运行
{
    printf("Hello from GPU\n"); // 在GPU上执行时，将这条消息打印到控制台
}

int main(void){ // C语言程序的主函数，从这里开始执行
    printf("Hello from CPU\n"); // 在CPU上执行时，将这条消息打印到控制台
    // 调用kernel核函数。
    // <<<1,1>>> 是CUDA的执行配置（execution configuration）。
    // 第一个1表示启动一个网格（grid），网格中包含1个块（block）。
    // 第二个1表示每个块中包含1个线程（thread）。
    // 也就是说，这个核函数将只在一个线程上执行。
    kernel<<<1,1>>>();
    hipDeviceSynchronize(); // 这是一个同步函数，强制CPU等待所有GPU上的操作完成。
                             // 这样做是为了确保在CPU程序结束前，GPU上的printf能够被执行并输出。
    return 0; // 主函数返回0，表示程序正常结束
}